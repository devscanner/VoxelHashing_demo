#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <glm/glm.hpp>
#include <stdio.h>

#include "CameraTracking.h"

using namespace glm;

//Since numCols = 640 and numRows = 480, we set blockDim according to 32x32 tile
dim3 blocks = dim3(20,15,1);
dim3 threads = dim3(32,32,1);

//Kinect v2 specific camera params
#define numCols 640
#define numRows 480
#define fx 525
#define fy 525
#define cx 319.5
#define cy 239.5

__global__
void calculateVertexPositions(vec4* d_vertexPositions, const uint16_t* d_depthBuffer)  {
  int xidx = blockDim.x*blockIdx.x + threadIdx.x;
  int yidx = blockDim.y*blockIdx.y + threadIdx.y;
  
  if(xidx >= numCols || yidx>= numRows) {
    return;
  }

  //find globalIdx
  int blockId = blockIdx.x + blockIdx.y * gridDim.x; 
  const int idx = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;


  float w = 1.0f; //flag to tell whether this is valid vertex or not
  uint16_t depth = d_depthBuffer[idx];
  //uint16_t lo = depth>>8;//d_depthBuffer[idx+1];
  //uint16_t hi = depth<<8;//d_depthBuffer[idx];
  //depth = hi+ lo;
  if(depth == 0) {
    w = 0.0f;
    //printf("validPixel %f at thread: %d\n", w, idx);
  }
  
  float x = ((xidx - cx)*depth)/fx;
  float y = ((yidx - cy)*depth)/fy;
  vec4 vertex = vec4(x, y, depth, w);
  if(idx<100){printf("thread: %d - %d\n", idx, depth);}
  d_vertexPositions[idx] = vertex;
}

__global__
void calculateNormals(vec4* d_positions, vec4* d_normals)
{
  int xidx = blockDim.x*blockIdx.x + threadIdx.x;
  int yidx = blockDim.y*blockIdx.y + threadIdx.y;
  
  if(xidx >= numCols || yidx>= numRows) {
    return;
  }

  
}

void CameraTracking::Align(vec4* d_input, vec4* d_inputNormals, vec4* d_target, 
  vec4* d_targetNormals, const uint16_t* d_depthInput, const uint16_t* d_depthTarget) {

  preProcess(d_input, d_inputNormals, d_depthInput);
  preProcess(d_target, d_targetNormals, d_depthTarget);

}

//Takes device pointers, calculates correct position and normals
void CameraTracking::preProcess(vec4 *positions, vec4* normals, const uint16_t *depth)  {
  calculateVertexPositions<<<blocks, threads>>>(positions, depth);
  calculateNormals<<<blocks, threads>>>(positions, normals);
}