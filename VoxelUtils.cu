#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "cuda_helper/hip/hip_vector_types.h"
#include "cuda_helper/hip/hip_runtime_api.h"
#include "VoxelDataStructures.h"
#include "common.h"

//This is a simple vector math library. Use this with CUDA instead of glm
#include "cuda_helper/cuda_SimpleMatrixUtil.h"

#define FREE_BLOCK -1
#define LOCKED_BLOCK -2
#define NO_OFFSET 0

__constant__ HashTableParams d_hashtableParams;
__device__ __constant__ float4x4 kinectProjectionMatrix;

VoxelEntry *d_hashTable;
VoxelEntry *d_compactifiedHashTable;
unsigned int *d_compactifiedHashCounter;
unsigned int *d_arena;	//Arena that manages free memory
unsigned int *d_arenaCounter;	//single element; points to next free block (atomic counter)
Voxel *d_voxelBlocks;
int *d_hashTableBucketMutex;	//mutex for locking particular bin while inserting/deleting


void updateConstantHashTableParams(const HashTableParams &params)	{
	size_t size;
	checkCudaErrors(hipGetSymbolSize(&size, HIP_SYMBOL(d_hashtableParams)));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(d_hashtableParams), &params, size, 0, hipMemcpyHostToDevice));
}

__host__
void allocate(const HashTableParams &params)	{
	checkCudaErrors(hipMalloc(&d_hashTable, sizeof(VoxelEntry) * params.numBuckets * params.bucketSize));
	checkCudaErrors(hipMalloc(&d_arena, sizeof(unsigned int) * params.numVoxelBlocks));
	checkCudaErrors(hipMalloc(&d_arenaCounter, sizeof(unsigned int)));
	checkCudaErrors(hipMalloc(&d_compactifiedHashTable, sizeof(VoxelEntry) * params.numBuckets * params.bucketSize));
	checkCudaErrors(hipMalloc(&d_voxelBlocks, sizeof(unsigned int) * params.numVoxelBlocks * params.voxelBlockSize * params.voxelBlockSize * params.voxelBlockSize));
	checkCudaErrors(hipMalloc(&d_hashTableBucketMutex, sizeof(int) * params.numBuckets));
}

__host__
void free()	{
	checkCudaErrors(hipFree(d_hashTable));
	checkCudaErrors(hipFree(d_arena));
	checkCudaErrors(hipFree(d_arenaCounter));
	checkCudaErrors(hipFree(d_compactifiedHashTable));
	checkCudaErrors(hipFree(d_compactifiedHashCounter));
	checkCudaErrors(hipFree(d_voxelBlocks));
	checkCudaErrors(hipFree(d_hashTableBucketMutex));
}

__host__
void calculateKinectProjectionMatrix()	{
	float m[4][4];
	m[0][0] = 2.0 * fx / imgWidth;
    m[0][1] = 0.0;
    m[0][2] = 0.0;
    m[0][3] = 0.0;

    m[1][0] = 0.0;
    m[1][1] = -2.0 * fy / imgHeight;
    m[1][2] = 0.0;
    m[1][3] = 0.0;

    m[2][0] = 1.0 - 2.0 * cx / imgWidth;
    m[2][1] = 2.0 * cy / imgHeight - 1.0;
    m[2][2] = (kinZFar + kinZNear) / (kinZNear - kinZFar);
    m[2][3] = -1.0;

    m[3][0] = 0.0;
    m[3][1] = 0.0;
    m[3][2] = 2.0 * kinZFar * kinZNear / (kinZNear - kinZFar);
    m[3][3] = 0.0;

	//Now upload to device
	cudaCheckErrors(hipMemcpyToSymbol(HIP_SYMBOL(kinectProjectionMatrix), m, sizeof(m)));
}


//Now actual GPU code
__device__
unsigned int calculateHash(const int3& pos)	{
		const int p0 = 73856093;
		const int p1 = 19349669;
		const int p2 = 83492791;

		int res = ((pos.x * p0) ^ (pos.y * p1) ^ (pos.z * p2)) % d_hashtableParams.numBuckets;
		if (res < 0) res += d_hashtableParams.numBuckets;
		return (uint)res;
}

__device__
float getTruncation(float z)	{
	return d_hashtableParams.truncation + (d_hashtableParams.truncScale * z);
}

__device__
int3 voxel2Block(int3 voxel) 	{
	const int size = d_hashtableParams.voxelBlockSize;
	if(voxel.x < 0) voxel.x -= size-1;	//i.e voxelBlockSize -1
	if(voxel.y < 0) voxel.y -= size-1;
	if(voxel.z < 0) voxel.z -= size-1;
	return make_int3(voxel.x/size, voxel.y/size, voxel.z/size);
}

__device__
int3 world2Voxel(const float3& point)	{
	const int size = d_hashtableParams.voxelBlockSize;
	float3 p = point/size;
	return make_int3(p + make_int3(signbit(p))*0.5);//return center
}

__device__
int3 block2Voxel(const int3& block)	{
	return block*d_hashtableParams.voxelBlockSize;
}

__device__
float3 voxel2World(const int3& voxel)	{
	return make_float3(voxel) * d_hashtableParams.voxelSize;
}

__device__
float3 block2World(const int3& block)	{
	return voxel2World(block2Voxel(block));
}

__device__
int3 world2Block(const float3& point)	{
	return voxel2Block(world2Voxel(point));
}

__device__
unsigned int linearizeVoxelPos(const int3& pos)	{
	const int size = d_hashtableParams.voxelBlockSize;
	return  pos.z * size * size +
			pos.y * size +
			pos.x;
}

__device__
int3 delinearizeVoxelPos(const unsigned int& index)	{
	const int size = d_hashtableParams.voxelBlockSize;
	unsigned int x = index % size;
	unsigned int y = (index % (size * size)) / size;
	unsigned int z = index / (size * size);
	return make_int3(x,y,z);
}


//what follows is IMO coolest code in project so far
__device__
VoxelEntry getVoxelEntry4Block(const int3& pos)	{
	const unsigned int hash = calculateHash(pos);
	const unsigned int bucketSize = d_hashtableParams.bucketSize;
	const unsigned int numBuckets = d_hashtableParams.numBuckets;
	const unsigned int startIndex = hash * bucketSize;

	VoxelEntry temp;
	temp.pos = pos;
	temp.offset = 0;
	temp.ptr = FREE_BLOCK;

	int i=0;
	//[1] Iterate all bucketSize entries
	for(i=0; i < bucketSize ; ++i)	{
		VoxelEntry& curr = d_hashTable[startIndex + i];
		if((curr.pos.x == pos.x) && (curr.pos.y == pos.y) &&(curr.pos.z == pos.z)
				&& (curr.ptr != FREE_BLOCK)) {
			return curr;
		}
	}
	//[2] block not found. handle collisions by traversing tail linked list
	const int lastEntryInBucket = (hash+1)*bucketSize -1;
	i = lastEntryInBucket;
	//memorize idx at list end and memorize offset from last
	//element of bucket to list end
	int iter = 0;
	const int maxIter = d_hashtableParams.attachedLinkedListSize;
	while(iter < maxIter)	{

		VoxelEntry curr = d_hashTable[i];
		if((curr.pos.x == pos.x) && (curr.pos.y == pos.y) &&(curr.pos.z == pos.z)
				&& (curr.ptr != FREE_BLOCK)) {
			return curr;
		}

		if(curr.offset == 0)	{ //we've found end of list
			break;
		}
		i = lastEntryInBucket + curr.offset;

		i %= (numBuckets * bucketSize);

		iter++;
	}
	return temp;
}


//TODO incomplete function
__device__
bool insertVoxelEntry(const int3& data)	{

	unsigned int hash = calculateHash(data);
	const unsigned int bucketSize = d_hashtableParams.bucketSize;
	const unsigned int numBuckets = d_hashtableParams.numBuckets;
	const unsigned int startIndex = hash * bucketSize;

	VoxelEntry temp;
	temp.offset=0;
	temp.ptr = FREE_BLOCK;
	temp.pos = data;

	//[1] iterate current bucket, try inserting at first empty block we see.
	int i=0;
	for(i=0; i<bucketSize; ++i)	{
		const int idx = startIndex+i;
		VoxelEntry &curr = d_hashTable[idx];
		if(curr.pos.x == data.x && curr.pos.y == data.y && curr.pos.z == data.z
				&& curr.ptr != FREE_BLOCK)	return false;
		if(curr.ptr == FREE_BLOCK)	{
			//TODO shouldn't the following be [hash] instead of [idx] ?
			int prevVal = atomicExch(&d_hashTableBucketMutex[hash], LOCKED_BLOCK);
			if(prevVal != LOCKED_BLOCK)	{	//means we can lock current bucket
				curr.pos = data;
				curr.offset = NO_OFFSET;
				curr.ptr = allocSingleBlockInHeap();
				return true;
			}
		}
	}
	//[2] bucket is full. Append to list.
	const int lastEntryInBucket = (hash+1)*bucketSize - 1;

	i = lastEntryInBucket;
	int offset=0;
	//memorize idx at list end and memorize offset from last
	//element of bucket to list end
	int iter = 0;
	const int maxIter = d_hashtableParams.attachedLinkedListSize;
	while(iter < maxIter)	{
		i = i%(numBuckets*bucketSize);
		VoxelEntry& curr = d_hashTable[i];
		if(curr.ptr != FREE_BLOCK)	{
			if(curr.pos.x == data.x && curr.pos.y == data.y &&
					curr.pos.z == data.z && curr.ptr != FREE_BLOCK)	{
				return false;	//alloc unsuccessful because block already there
			}
			if(curr.offset == 0)	{//end of list, lookahead till we find empty slot
				int j=1;
				//[1] lock the parent block
				int prevVal = atomicExch(&d_hashTableBucketMutex[hash],
						LOCKED_BLOCK);
				if(prevVal != LOCKED_BLOCK)	{//if we got the lock
					//[2] then lookahead for empty block in new bucket
					while(j<10)	{
						if(d_hashTable[i+j].ptr == FREE_BLOCK)	break;
						j++;
					}
					if(j==10)	{
						//we couldn't find empty space despite looking ahead 10 spaces
						return false;
					}
					//[3] now lock this new bucket and insert the block
					prevVal = atomicExch(&d_hashTableBucketMutex[(i+j)/numBuckets],
							LOCKED_BLOCK);
					if(prevVal != LOCKED_BLOCK)	{
						VoxelEntry& next = d_hashTable[i+j];
						//TODO maybe we can do away with this check
						if(next.ptr == FREE_BLOCK)	{
							next.ptr = allocSingleBlockInHeap();
							next.pos = data;
							curr.offset = j;
							break;
						}
						i++;
					}
				}
			}
			//TODO: implement this correctly
			if(curr.offset != 0)	{	//traversing nodes in linked list
				int j = i;
				while(j <= (i+curr.offset))	{
					if(d_hashTable[j].ptr == FREE_BLOCK)	{
						//[a] free space found. first lock bucket with curr
						int prevVal = atomicExch(&d_hashTableBucketMutex[hash/numBuckets], LOCKED_BLOCK);
						if(prevVal != LOCKED_BLOCK)	{
							//[b] then lock bucket with new space
							prevVal = atomicExch(&d_hashTableBucketMutex[j/numBuckets], LOCKED_BLOCK);
							if(prevVal != LOCKED_BLOCK)	{
								VoxelEntry& ins = d_hashTable[j];
								ins.offset = i + curr.offset - j;
								ins.ptr = allocSingleBlockInHeap();
								ins.pos = data;
								curr.offset = j - i;
								return true;
							}
						}
					}
					j++;
				}
				i += curr.offset;
			}
		}
		iter++;
	}
}


__device__
int beforeThis(int3 data)	{

	unsigned int hash = calculateHash(data);
	const unsigned int bucketSize = d_hashtableParams.bucketSize;
	const unsigned int numBuckets = d_hashtableParams.numBuckets;
	const unsigned int startIndex = hash * bucketSize;

	int iter = 0; const int maxiter = 7;
	int i = startIndex;
	while(iter < maxIter)	{
		const VoxelEntry& curr = d_hashTable[i];
		const VoxelEntry& next = d_hashTable[i + curr.offset];
		if((next.pos.x==data.x) && (next.pos.y==data.y) && (next.pos.z==data.z))	{
			return i;
		}
		i += curr.offset;
		iter++;
	}
	return -1;	//error; should not happen
}


__device__
void removeSingleBlockInHeap(int ptr)	{
	int delIdx = ptr / 512;
	uint addr = atomicSub(&d_freeMemoryCounter, 1);
	d_heap[addr + 1] = ptr;
}

__device__
void deleteVoxelEntry(int3 data)	{
	//TODO : iterate over entire bucket
	unsigned int hash = calculateHash(data);
	const unsigned int bucketSize = d_hashtableParams.bucketSize;
	const unsigned int numBuckets = d_hashtableParams.numBuckets;
	const unsigned int startIndex = hash * bucketSize;

	VoxelEntry temp;
	temp.offset=0;
	temp.ptr = FREE_BLOCK;
	temp.pos = data;

	//[1] iterate current bucket, try inserting at first empty block we see.
	int i=0;
	for(i=0; i<bucketSize; ++i)	{
		const int idx = startIndex+i;
		VoxelEntry &curr = d_hashTable[idx];
		if(curr.pos.x == data.x && curr.pos.y == data.y && curr.pos.z == data.z
				&& curr.ptr != FREE_BLOCK)	{return false;}
		if(curr.ptr == FREE_BLOCK)	{
			//TODO shouldn't the following be [hash] instead of [idx] ?
			//try locking current bucket
			int prevVal = atomicExch(&d_hashTableBucketMutex[hash], LOCKED_BLOCK);
			if(prevVal != LOCKED_BLOCK)	{	//means we can lock current bucket
				curr.pos = make_int3(0);
				curr.offset = NO_OFFSET;
				removeSingleBlockInHeap(curr.ptr);
				curr.ptr = FREE_BLOCK;
				return true;
			}
		}
	}
	//deletion in linked list
	int lastEntry = beforeThis(data);
	if(lastEntry == -1)	{return;}	//error
	VoxelEntry& prev = d_hashTable[lastEntry];
	VoxelEntry& curr = d_hashTable[lastEntry + prev.offset];
	//lock the bucket

}

__device__
void allocBlocksKernel(const float4* verts, const float4* normals)	{	//Do we need normal data here?

	const int voxSize = d_hashtableParams.voxelSize;
	int xidx = blockDim.x*blockIdx.x + threadIdx.x;
	int yidx = blockDim.y*blockIdx.y + threadIdx.y;

	if (xidx >= numCols || yidx >= numRows) {
		return;
	}

	//find globalIdx row-major
	const int idx = (yidx*numCols) + xidx;

	float3 p = make_float3(verts[idx]);
	float3 pn = make_float3(normals[idx]);
	float3 rayStart = p - (d_hashtableParams.truncation * pn);
	float3 rayEnd = p + (d_hashtableParams.truncation * pn);

	//Now find their voxel blocks
	int3 startBlock = world2Block(rayStart);
	int3 endBlock = world2Block(rayEnd);
	float3 rayDir = normalize(rayEnd - rayStart);

	int3 step = make_int3(signbit(rayDir));	//block stepping size
	float3 next_boundary = (rayStart + make_float3(step));

	//calculate distance to next barrier
	float3 tMax = (make_float3(next_boundary - rayStart)) / rayDir;
	float3 tDelta = (voxSize / rayDir);
	tDelta *= step;

	//convert to voxel-blocks
	int3 idStart = world2Block(rayStart);
	int3 idEnd = world2Block(rayEnd);
	int3 temp = idStart;

	if (rayDir.x == 0.0f) { tMax.x = INF; tDelta.x = INF; }
	if (next_boundary.x - rayStart.x == 0.0f) { tMax.x = INF; tDelta.x = INF; }

	if (rayDir.y == 0.0f) { tMax.y = INF; tDelta.y = INF; }
	if (next_boundary.y - rayStart.y == 0.0f) { tMax.y = INF; tDelta.y = INF; }

	if (rayDir.z == 0.0f) { tMax.z = INF; tDelta.z = INF; }
	if (next_boundary.z - rayStart.z == 0.0f) { tMax.z = INF; tDelta.z = INF; }

	//first insert idStart block into the hashtable
	insertVoxelEntry(temp);

	while(temp != idEnd)	{
		if(tMax.x < tMax.y && tMax.x < tMax.z)	{
			temp.x += step.x;
			//if(temp.x == idEnd.x) break;
			tMax.x += tDelta.x;
		}
		else if(tMax.z < tMax.y)	{
			temp.z += step.z;
			//if(temp.z == idEnd.z) break;
			tMax.z += tDelta.z;
		}
		else{
			temp.y += step.y;
			//if(temp.y == idEnd.y) break;
			tMax.y += tDelta.y;
		}

		//check if block is in view, then insert into table
		if(blockInFrustum(temp))	{
			insertVoxelEntry(temp);
		}
		//cout<<"\nVisited "<<glm::to_string(temp);
		iter++;
	}
	//By now all necessary blocks will have been allocated
}

__inline__ __device__
bool blockInFrustum(const int3& blockId)	{
	float4 pos = make_float4(blockId);
	pos = d_hashtableParams.global_transform * pos;
	pos = kinectProjectionMatrix * pos;

	if((pos.x > -pos.w) && (pos.x < pos.w) &&
		(pos.y > -pos.w) && (pos.y < pos.w) &&
		(pos.z > -pos.w) && (pos.z < pos.w))	{
		return true;
	}
	else {
		return false;
	}
}

