#include "hip/hip_runtime.h"
#ifndef CAMTRACKING_UTIL
#define CAMTRACKING_UTIL

#if defined(_WIN32)
  #include <Windows.h>
#endif

#include <iostream>
#include <hip/hip_runtime_api.h>
#include "cuda_helper/hip/hip_runtime_api.h"
#include "cuda_helper/hip/hip_vector_types.h"
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>

//This is a simple vector library. Use this with CUDA instead of GLM.
#include "cuda_helper/cuda_SimpleMatrixUtil.h"

//#define MINF __int_as_float(0xff800000)
//#define MAXF __int_as_float(0x7F7FFFFF)
#define fx 525
#define fy 525
#define cx 319.5
#define cy 239.5
//Kinect v2 specific camera params
#define numCols 640
#define numRows 480

const float distThres = 5.0f;
const float normalThres = -1.0f;
const float idealError = 0.0f;
//Since numCols = 640 and numRows = 480, we set blockDim according to 32x32 tile
dim3 blocks = dim3(20, 15, 1);
dim3 threads = dim3(32, 32, 1);

__device__ __constant__ float3x3 K;  //Camera intrinsic matrix
__device__ __constant__ float3x3 K_inv;
__device__ float globalError;

__device__ inline
bool isValid(float4 v) {
	return v.w != MINF;
}

__device__
static inline int2 cam2screenPos(float3 p) {
  float3 sp = K*p;
  //return make_int2(sp.x + 0.5, sp.y + 0.5);
	//float x = ((p.x * fx) / p.z) + cx;
	//float y = ((p.y * fy) / p.z) + cy;
	return make_int2(sp.x/sp.z + 0.5, sp.y/sp.z + 0.5);
}

__global__
void calculateVertexPositions(float4* d_vertexPositions, const uint16_t* d_depthBuffer) {
	int xidx = blockDim.x*blockIdx.x + threadIdx.x;
	int yidx = blockDim.y*blockIdx.y + threadIdx.y;

	if (xidx >= numCols || yidx >= numRows) {
		return;
	}

	//find globalIdx row-major
	const int idx = (yidx*numCols) + xidx;

	float w = 1.0f; //flag to tell whether this is valid vertex or not
	uint16_t d = d_depthBuffer[idx];
	float depth = d / 5000.0f; //5000 units = 1meter. We're now dealing in meters.
	if (depth == 0) {
		w = 0.0f;
	}

  float3 imageCoord = make_float3(xidx, yidx, 1.0);
  float3 point = K_inv*imageCoord*depth;
  float4 vertex = make_float4(point.x, -point.y, -point.z, w);
  d_vertexPositions[idx] = vertex;
}

__global__
void calculateNormals(const float4* d_positions, float4* d_normals)
{
	int xidx = blockDim.x*blockIdx.x + threadIdx.x;
	int yidx = blockDim.y*blockIdx.y + threadIdx.y;

	if (xidx >= numCols || yidx >= numRows) {
		return;
	}

	//find globalIdx row-major
	const int idx = (yidx*numCols) + xidx;

	d_normals[idx] = make_float4(MINF, MINF, MINF, MINF);

	if (xidx > 0 && xidx < numCols - 1 && yidx > 0 && yidx < numRows - 1) {
		const float4 CC = d_positions[(yidx + 0)*numCols + (xidx + 0)];
		const float4 PC = d_positions[(yidx + 1)*numCols + (xidx + 0)];
		const float4 CP = d_positions[(yidx + 0)*numCols + (xidx + 1)];
		const float4 MC = d_positions[(yidx - 1)*numCols + (xidx + 0)];
		const float4 CM = d_positions[(yidx + 0)*numCols + (xidx - 1)];

		if (CC.x != MINF && PC.x != MINF && CP.x != MINF && MC.x != MINF && CM.x != MINF)
		{
			const float3 n = cross(make_float3(PC) - make_float3(MC), make_float3(CP) - make_float3(CM));
			const float  l = length(n);

			if (l > 0.0f)
			{
				//float4 v = make_float4(n/-l, 1.0f);
				float4 vert = make_float4(n/l, 1.0);
				d_normals[idx] = vert;
				//printf("Normal for thread %d : %f %f %f", yidx*numRows+xidx, vert.x, vert.y, vert.z);
			}
		}
	}
}

extern "C" void preProcess(float4 *positions, float4* normals, const uint16_t *depth) {
	calculateVertexPositions <<<blocks, threads >>>(positions, depth);
	calculateNormals <<<blocks, threads >>>(positions, normals);
	checkCudaErrors(hipDeviceSynchronize());

}


__global__
void FindCorrespondences(const float4* input,	const float4* target,
    const float4* targetnormals, float4* correspondences, float4* correspondenceNormals,, float* residuals,	const float4x4 deltaT,
    float distThres, float normalThres, int width, int height)
{

  const int offset = 1;
	int xidx = blockDim.x*blockIdx.x + threadIdx.x;
	int yidx = blockDim.y*blockIdx.y + threadIdx.y;

	if (xidx >= numCols || yidx >= numRows) {
		return;
	}

	const int idx = (yidx*width) + xidx;

	float4 pSrc = input[idx];

	if (pSrc.z != 0) {	//if both pos and normal are valid points
    pSrc.w = 1.0f;
    float4 transPSrc = deltaT * pSrc;

		int2 projected = cam2screenPos(make_float3(transformedPos));
    int2 &sp = projected;
    sp /= offset;

    if(sp.x > 0 && sp.y > 0 && sp.x < width && sp.y < height)
    {
      int targetIndex = (sp.y * width) + sp.x;
      float4 pTar = target[targetIndex];
      float4 nTar = targetNormals[targetIndex];
      float3 diff = make_float3(transPSrc - pTar);
      float d = dot(diff, make_float3(nTar));
      if (d < distThres)  {
        atomicAdd(&globalError, d);
        correspondences[idx] = pTar;
        correspondenceNormals[idx] = nTar;
        residuals[idx] = d;
        //coordpairs[idx].srcindex = idx;
        //coordpairs[idx].targIndex = targetIndex;
        //coordpairs[idx].srcindex = d;
      }
    }
	}
}

extern "C" float computeCorrespondences(const float4* d_input, const float4* d_target,
    const float4* d_targetNormals, thrust::device_vector<float4>& corres,
    thrust::device_vector<float4>& corresNormals,thrust::device_vector<float>& residual,
    const float4x4 deltaTransform, const int width, const int height)
{
	//First clear the previous correspondence calculation
  CoordPair temp;
  checkCudaErrors(thrust::fill(coordPairs.begin(), coordPairs.end(), temp));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(globalError), &idealError, sizeof(float)));

	FindCorrespondences <<<blocks, threads>>>(d_input, d_target, d_targetNormals,
      d_correspondences, d_corresNormals, d_residuals,	deltaTransform, distThres, normalThres, width, height);

  float globalErrorReadback = 0.0;
  checkCudaErrors(hipMemcpyFromSymbol(&globalErrorReadback, HIP_SYMBOL(globalError), sizeof(float)));
  //std::cout<<"Global correspondence error = "<<globalErrorReadback<<" \n\n";
	checkCudaErrors(hipDeviceSynchronize());
  return globalErrorReadback;
}

extern "C" bool SetCameraIntrinsic(const float* intrinsic, const float* invIntrinsic) {
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(K), intrinsic, 9*sizeof(float)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(K_inv), invIntrinsic, 9*sizeof(float)));
  return true;
}
#endif // CAMTRACKING_UTIL
