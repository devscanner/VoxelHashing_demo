#include "hip/hip_runtime.h"
#ifndef CAMTRACKING_UTIL
#define CAMTRACKING_UTIL

#if defined(_WIN32)
  #include <Windows.h>
#endif

#include <iostream>
#include <cstdio>
#include <hip/hip_runtime_api.h>
#include "cuda_helper/hip/hip_runtime_api.h"
#include "cuda_helper/hip/hip_vector_types.h"
//#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include "common.h"

//This is a simple vector library. Use this with CUDA instead of GLM.
#include "cuda_helper/cuda_SimpleMatrixUtil.h"

//#define MINF __int_as_float(0xff800000)
//#define MAXF __int_as_float(0x7F7FFFFF)
#define fx 525
#define fy 525
#define cx 319.5
#define cy 239.5
//Kinect v2 specific camera params
#define numCols 640
#define numRows 480

//const float distThres = 5.0f;
//const float normalThres = -1.0f;
//const float idealError = 0.0f;
//Since numCols = 640 and numRows = 480, we set blockDim according to 32x32 tile
dim3 blocks = dim3(20, 15, 1);
dim3 threads = dim3(32, 32, 1);

//using thrust::device_vector;
//using thrust::device_ptr;

__device__ __constant__ float3x3 K;  //Camera intrinsic matrix
__device__ __constant__ float3x3 K_inv;
__device__ float globalError;

//__device__ inline
//bool isValid(float4 v) {
//	return v.w != MINF;
//}

__global__
void calculateVertexPositions(float4* d_vertexPositions, const uint16_t* d_depthBuffer) {
	int xidx = blockDim.x*blockIdx.x + threadIdx.x;
	int yidx = blockDim.y*blockIdx.y + threadIdx.y;

	if (xidx >= numCols || yidx >= numRows) {
		return;
	}

	//find globalIdx row-major
	const int idx = (yidx*numCols) + xidx;

	const float w = 1.0f; //flag to tell whether this is valid vertex or not
	uint16_t d = d_depthBuffer[idx];
	float depth = d / 5000.0f; //5000 units = 1meter. We're now dealing in meters.
	//if (depth == 0) {
	//	w = 0.0f;
	//}

  float3 imageCoord = make_float3(xidx, yidx, 1.0);
  float3 point = K_inv*imageCoord*depth;
  //float4 vertex = make_float4(point.x, -point.y, -point.z, w);
  float4 vertex = make_float4(point.x, point.y, point.z, w);
  d_vertexPositions[idx] = vertex;
}

__global__
void calculateNormals(const float4* d_positions, float4* d_normals)
{
	int xidx = blockDim.x*blockIdx.x + threadIdx.x;
	int yidx = blockDim.y*blockIdx.y + threadIdx.y;

	if (xidx >= numCols || yidx >= numRows) {
		return;
	}

	//find globalIdx row-major
	const int idx = (yidx*numCols) + xidx;

	//d_normals[idx] = make_float4(MINF, MINF, MINF, MINF);
	d_normals[idx] = make_float4(0, 0, 0, 0);

	if (xidx > 0 && xidx < numCols - 1 && yidx > 0 && yidx < numRows - 1) {
		const float4 CC = d_positions[(yidx + 0)*numCols + (xidx + 0)];
		const float4 PC = d_positions[(yidx + 1)*numCols + (xidx + 0)];
		const float4 CP = d_positions[(yidx + 0)*numCols + (xidx + 1)];
		const float4 MC = d_positions[(yidx - 1)*numCols + (xidx + 0)];
		const float4 CM = d_positions[(yidx + 0)*numCols + (xidx - 1)];

		if (CC.x != 0 && PC.x != 0 && CP.x != 0 && MC.x != 0 && CM.x != 0)
		{
			const float3 n = cross(make_float3(PC) - make_float3(MC), make_float3(CP) - make_float3(CM));
			const float  l = length(n);

			if (l > 0.0f)
			{
				//float4 v = make_float4(n/-l, 1.0f);
				float4 vert = make_float4(n/l, 0.0);
				d_normals[idx] = vert;
				//printf("Normal for thread %d : %f %f %f", yidx*numRows+xidx, vert.x, vert.y, vert.z);
			}
		}
	}
}

extern "C" void preProcess(float4 *positions, float4* normals, const uint16_t *depth) {
	calculateVertexPositions <<<blocks, threads >>>(positions, depth);
	calculateNormals <<<blocks, threads >>>(positions, normals);
	checkCudaErrors(hipDeviceSynchronize());

}

__device__
static inline int2 cam2screenPos(float3 p) {
  float3 sp = K*p;
  //return make_int2(sp.x + 0.5, sp.y + 0.5);
	//float x = ((p.x * fx) / p.z) + cx;
	//float y = ((p.y * fy) / p.z) + cy;
	return make_int2(sp.x/sp.z + 0.5, sp.y/sp.z + 0.5);
}

__global__
void FindCorrespondences(const float4* input,	const float4* target,
    const float4* targetNormals, float4* correspondences, float4* correspondenceNormals,
    float* residuals,	const float4x4 deltaT,
    float distThres, float normalThres, int width, int height)
{

  const int offset = 1;
	int xidx = blockDim.x*blockIdx.x + threadIdx.x;
	int yidx = blockDim.y*blockIdx.y + threadIdx.y;

  //if (threadIdx.x==0 && threadIdx.y ==0)  {
  //  printf("Block is (%i, %i)\n",blockIdx.x, blockIdx.y);
  //}
	if (xidx >= numCols || yidx >= numRows) {
		return;
	}

	const int idx = (yidx*width) + xidx;

	float4 pSrc = input[idx];

	if (pSrc.z != 0) {	//if both pos and normal are valid points
    pSrc.w = 1.0f;
    float4 transPSrc = deltaT * pSrc;

		int2 projected = cam2screenPos(make_float3(transPSrc));
    int2 &sp = projected;
    //sp.x = sp.x/offset;
    //sp.y = sp.y/offset;

    if(sp.x > 0 && sp.y > 0 && sp.x < width && sp.y < height)
    {
      //printf("%i) sp.x = %i
      int targetIndex = (sp.y * width) + sp.x;
      float4 pTar = target[targetIndex];
      float4 nTar = targetNormals[targetIndex];
      float3 diff = make_float3(transPSrc - pTar);
      float d = dot(diff, make_float3(nTar));
      if (d < distThres)  {
        //if (threadIdx.x ==0 && threadIdx.y ==0)
        {
          //printf("%i) src- (%f, %f, %f), target- (%f, %f, %f), d= %f\n",idx, pSrc.x, pSrc.y, pSrc.z, pTar.x, pTar.y, pTar.z, d);
        }
        atomicAdd(&globalError, d);
        correspondences[idx] = pTar;
        correspondenceNormals[idx] = nTar;
        residuals[idx] = d;
        //coordpairs[idx].srcindex = idx;
        //coordpairs[idx].targIndex = targetIndex;
        //coordpairs[idx].srcindex = d;
      }
    }
	}
}

extern "C" float computeCorrespondences(const float4* d_input, const float4* d_target,
    const float4* d_targetNormals, float4* corres,
    float4* corresNormals, float* residuals,
    const float4x4 deltaTransform, const int width, const int height)
{
	//First clear the previous correspondence calculation
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(globalError), &idealError, sizeof(float)));

  thrust::device_ptr<float4> corres_ptr = thrust::device_pointer_cast(corres);
  thrust::device_ptr<float4> corresNormals_ptr = thrust::device_pointer_cast(corresNormals);
  thrust::device_ptr<float> residuals_ptr = thrust::device_pointer_cast(residuals);

  //std::cerr<<"Before clearing prev correspondences\n";

  thrust::fill(corres_ptr, corres_ptr + (width*height), float4{0,0,0,0});
  thrust::fill(corresNormals_ptr, corresNormals_ptr+ (width*height), float4{0,0,0,0});
  thrust::fill(residuals_ptr, residuals_ptr+ (width*height), (float)0.0f);

  checkCudaErrors(hipDeviceSynchronize());
  //std::cerr<<"After clearing prev correspondences\n";
	FindCorrespondences <<<blocks, threads>>>(d_input, d_target, d_targetNormals,
      corres, corresNormals, residuals,	deltaTransform, distThres, normalThres, width, height);
  checkCudaErrors(hipDeviceSynchronize());

  float globalErrorReadback = 0.0;
  checkCudaErrors(hipMemcpyFromSymbol(&globalErrorReadback, HIP_SYMBOL(globalError), sizeof(float)));
  //std::cerr<<"Global correspondence error = "<<globalErrorReadback<<" \n\n";
	checkCudaErrors(hipDeviceSynchronize());
  return globalErrorReadback;
}

extern "C" bool SetCameraIntrinsic(const float* intrinsic, const float* invIntrinsic) {
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(K), intrinsic, 9*sizeof(float)));
  checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(K_inv), invIntrinsic, 9*sizeof(float)));
  return true;
}
#endif // CAMTRACKING_UTIL
